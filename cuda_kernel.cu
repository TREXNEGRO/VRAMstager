#include <hip/hip_runtime.h>

__global__ void xor_decrypt_kernel(unsigned char* input, unsigned char* output, unsigned char key, size_t size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        output[idx] = input[idx] ^ key;
    }
}
